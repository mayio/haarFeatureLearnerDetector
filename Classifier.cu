#include "hip/hip_runtime.h"
/*
 *
 *  Created on: May 17, 2017
 *      Author: Mario Lüder
 *
 *
 */

#include "Classifier.cuh"
#include "defines.cuh"
#include "GpuStrongClassifier.cuh"
#include "utilities.cuh"

#include <opencv2/core/gpumat.hpp>

#include <iostream>

// texture
texture<int32_t, 2, hipReadModeElementType> texIntegralImage;

__forceinline__ __device__ void Classifier::getFeatureValueTex(
      const uint8_t * const classifier,
      const uint32_t x,
      uint32_t y,
      const uint32_t rectElementWidth, const uint32_t rectElementHeight,
      const uint32_t featureWidth, const uint32_t featureHeight,
      int32_t & value)
{
   assert(false);

   // FIXME:
   // this is not right
   // see getFeatureValue
   // the rectElementWidth and rectElementHeight have to be reduced by 1
   //

   value = 0;
   uint32_t xi = x;

   uint32_t y2 = y + rectElementHeight;
   uint32_t x2 = x + rectElementWidth;

   int32_t i1 = tex2D(texIntegralImage, x,  y); //  integralImageLine[x];                     // integralImage(yi, xi);
   int32_t i2 = tex2D(texIntegralImage, x2, y); //  integralImageLine[x + rectElementWidth];  // integralImage(yi, xi + rectElementWidth);
   int32_t i3 = tex2D(texIntegralImage, x,  y2); // integralImageLine2[x];                    //integralImage(yi + rectElementHeight, xi);
   int32_t i4 = tex2D(texIntegralImage, x2, y2); //integralImageLine2[x + rectElementWidth]; //integralImage(yi + rectElementHeight, xi + rectElementWidth);

   int32_t k3 = i3;
   int32_t k4 = i4;

   for (uint32_t h = 0; h < featureHeight; ++h)
   {
      for (uint32_t w = 0; w < featureWidth; ++w)
      {
         int32_t rectangleType = 0;
         Classifier::getRectangleType(classifier, h * featureWidth + w, rectangleType);
         value += ((i4 + i1 - i2 - i3) * rectangleType);

         if (w + 1 < featureWidth)
         {
            i1 = i2;
            i3 = i4;

            xi += rectElementWidth;
            const uint32_t xi2 = xi + rectElementWidth;
            i2 = tex2D(texIntegralImage, xi2, y); // integralImageLine[xi2];
            i4 = tex2D(texIntegralImage, xi2, y2); // integralImageLine2[xi2];
         }
      }

      if (h + 1 < featureHeight)
      {
         xi = x;
         y = y2; // integralImageLine = integralImageLine2;
         y2 += rectElementHeight; // integralImageLine2 = (int32_t*)((uint8_t*)(integralImageLine2) + rectElementHeight * lineStep);
         i1 = k3;
         i2 = k4;
         i3 = tex2D(texIntegralImage, xi, y2); // integralImageLine2[xi];
         i4 = tex2D(texIntegralImage, xi + rectElementWidth, y2); // integralImageLine2[xi + rectElementWidth];

         k3 = i3;
         k4 = i4;
      }
   }
}

std::string Classifier::dumpSelectedClassifier(
      const Classifier::SelectionResult & selected,
      const FeatureTypes & featureTypes)
{
   std::stringstream prettyClassifier;
   uint32_t maxFeatureTypes;

   Classifier::getClassifierTypesCount(featureTypes.getData(), maxFeatureTypes);
   assert(maxFeatureTypes > selected.classifierTypeIdx);

   uint32_t featureWidth;
   uint32_t featureHeight;
   uint32_t maxClassifier;
   const uint8_t * allClassifier = NULL;
   const uint8_t * selectedClassifierPtr = NULL;

   Classifier::getClassifier(featureTypes.getData(), selected.classifierTypeIdx, maxClassifier, &allClassifier,
         featureWidth, featureHeight);
   assert(maxClassifier > selected.classifierIdx);
   assert(allClassifier);

   uint32_t rectWidth;
   uint32_t rectHeight;

   Classifier::getClassifierScale(allClassifier, selected.classifierIdx, &selectedClassifierPtr,
         rectWidth, rectHeight);
   assert(selectedClassifierPtr);

   prettyClassifier << "[";
   prettyClassifier << "[";
   for (uint32_t i = 0; i < featureHeight; ++i)
   {
      prettyClassifier << "[";

      for (uint32_t j = 0; j < featureWidth; ++j)
      {
         prettyClassifier << "[";
         prettyClassifier << rectWidth << ",";
         prettyClassifier << rectHeight << ",";
         int32_t type;
         Classifier::getRectangleType(selectedClassifierPtr, i * featureWidth + j, type);
         prettyClassifier << type;
         prettyClassifier << "]";

         if ((j + 1) < featureWidth)
         {
            prettyClassifier << ",";
         }
      }

      prettyClassifier << "]";

      if ((i + 1) < featureHeight)
      {
         prettyClassifier << ",";
      }
   }
   prettyClassifier << "]";
   prettyClassifier << "," << selected.x;
   prettyClassifier << "," << selected.y;
   prettyClassifier << "," << selected.error;
   prettyClassifier << "," << selected.threshold;
   prettyClassifier << "," << selected.polarity;
   prettyClassifier << "]";

   return prettyClassifier.str();
}

bool Classifier::fromResult(const std::string & result, std::vector<Classifier::Stage> & strongClassifier, FeatureTypes & featureTypes)
{
   // [[ [[[[[49,22,1]],
   //    [[[[[
   const size_t strLength = result.size();

   if (strLength == 0)
   {
      return false;
   }

   std::string::size_type pos = result.find_first_of("[");
   std::string::size_type lastPos = pos;

   if (pos == std::string::npos)
   {
      return false;
   }

   pos++;
   lastPos++;
   const char * const delimiter = "[],";
   std::cout << "[";

   pos = result.find_first_of(delimiter, pos);
   while(pos != std::string::npos && result[pos] != ']')
   {
      // stage
      Classifier::Stage stage;

      if (pos != std::string::npos && result[pos] == ',')
      {
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
         std::cout << ",";
      }

      if (pos != std::string::npos)
      {
         pos++;
         pos = result.find_first_of(delimiter, pos);
      }

      if (pos != std::string::npos && result[pos] == '[')
      {
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
         std::cout << "[";
      }
      else
      {
         std::cout << " Unexpected delimiter. Expected ','" << std::endl;
         return false;
      }

      // [[[[[19,16,1]],[[19,16,-1]]],12,6,0,-12641,1],
      while(pos != std::string::npos && result[pos] != ']')
      {
         if (pos != std::string::npos && result[pos] == ',')
         {
            pos++;
            lastPos = pos;
            pos = result.find_first_of(delimiter,pos);
            std::cout << ",";
         }

         if (pos != std::string::npos && result[pos] == '[')
         {
            // Classifier with parameter
            std::cout << "[";
            pos++;
            pos = result.find_first_of(delimiter,pos);

            uint32_t width = 0;
            uint32_t height = 0;

            FeatureType feature(0,0);
            Classifier::SelectionResult selectionResult;
            selectionResult.classifierIdx = 0;
            selectionResult.classifierTypeIdx = featureTypes.size();

            if (pos != std::string::npos && result[pos] == '[')
            {
               // Classifier
               //[[[19,16,1]],[[19,16,-1]]]
               std::cout << "[";
               pos++;
               pos = result.find_first_of(delimiter,pos);

               while(pos != std::string::npos && result[pos] != ']')
               {
                  if (pos != std::string::npos && result[pos] == ',')
                  {
                     pos++;
                     lastPos = pos;
                     pos = result.find_first_of(delimiter,pos);
                     std::cout << ",";
                  }

                  if (pos != std::string::npos && result[pos] == '[')
                  {
                     // Row
                     feature.addRow();
                     std::cout << "[";
                     pos++;
                     pos = result.find_first_of(delimiter,pos);

                     while(pos != std::string::npos && result[pos] != ']')
                     {
                        if (pos != std::string::npos && result[pos] == ',')
                        {
                           pos++;
                           lastPos = pos;
                           pos = result.find_first_of(delimiter,pos);
                           std::cout << ",";
                        }

                        if (pos != std::string::npos && result[pos] == '[')
                        {
                           // Column
                           std::cout << "[";

                           if (parseValue(result, delimiter, ",", pos, lastPos, width))
                           {
                              std::cout << width << ",";
                           }

                           if (parseValue(result, delimiter, ",", pos, lastPos, height))
                           {
                              std::cout << height << ",";
                           }

                           int32_t type;

                           if (parseValue(result, delimiter, "]", pos, lastPos, type))
                           {
                              std::cout << type << "]";
                           }

                           feature << type;
                        }

                        if (pos != std::string::npos)
                        {
                           pos++;
                           lastPos = pos;
                           pos = result.find_first_of(delimiter,pos);
                        }
                     }
                     std::cout << "]";
                  }

                  if (pos != std::string::npos)
                  {
                     pos++;
                     lastPos = pos;
                     pos = result.find_first_of(delimiter,pos);
                  }
               }

               if (pos != std::string::npos)
               {
                  pos++;
                  lastPos = pos;
                  pos = result.find_first_of(delimiter,pos);
               }

               if (parseValue(result, delimiter, ",", pos, lastPos, selectionResult.x))
               {
                   std::cout << "]," << selectionResult.x << ",";
               }

               if (parseValue(result, delimiter, ",", pos, lastPos, selectionResult.y))
               {
                   std::cout << selectionResult.y << ",";
               }

               if (parseValue(result, delimiter, ",", pos, lastPos, selectionResult.error))
               {
                   std::cout << selectionResult.error << ",";
               }

               if (parseValue(result, delimiter, ",", pos, lastPos, selectionResult.threshold))
               {
                   std::cout << selectionResult.threshold << ",";
               }

               if (parseValue(result, delimiter, "]", pos, lastPos, selectionResult.polarity))
               {
                   std::cout << selectionResult.polarity << "]";
               }
            }

            feature.setRect(width, height);
            featureTypes.push_back(feature);
            stage.stagedClassifier.push_back(selectionResult);
         }

         if (pos != std::string::npos)
         {
            pos++;
            lastPos = pos;
            pos = result.find_first_of(delimiter,pos);
         }
      }

      // ,[0.0343905,0.110103],2.20634]
      //
      // betas
      //
      if (pos != std::string::npos)
      {
         std::cout << result[pos];
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
      }

      if (pos != std::string::npos && result[pos] == ',')
      {
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
         std::cout << ",";
      }
      else
      {
         std::cout << " Unexpected delimiter. Expected ','" << std::endl;
         return false;
      }

      if (pos != std::string::npos)
      {
         std::cout << result[pos];
      }

      while(pos != std::string::npos && result[pos] != ']')
      {
         double beta;

         if (parseValue(result, delimiter, ",]", pos, lastPos, beta))
         {
            stage.betas.push_back(beta);
            std::cout << beta << result[pos];
         }
      }

      // stage threshold

      if (pos != std::string::npos)
      {
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
      }

      if (pos != std::string::npos && result[pos] == ',')
      {
         std::cout << ",";
      }
      else
      {
         std::cout << " Unexpected delimiter. Expected ','" << std::endl;
         return false;
      }

      if (parseValue(result, delimiter, "]", pos, lastPos, stage.stageThreshold))
      {
         std::cout << stage.stageThreshold << result[pos];
      }

      strongClassifier.push_back(stage);

      if (pos != std::string::npos)
      {
         pos++;
         lastPos = pos;
         pos = result.find_first_of(delimiter,pos);
      }
      // stage end
      std::cout << std::endl;
   }

   std::cout << "]" << std::endl;
   return true;
}

void Classifier::sizeStrongClassifier(
         const std::vector<Classifier::Stage> & strongClassifier,
         const FeatureTypes & featureTypes,
         uint32_t & xMin,
         uint32_t & yMin,
         uint32_t & xMax,
         uint32_t & yMax)
{

   xMax = 0;
   yMax = 0;
   xMin = INT32_MAX;
   yMin = INT32_MAX;

   for (std::vector<Classifier::Stage>::const_iterator stageIter = strongClassifier.begin();
        stageIter != strongClassifier.end();
        ++stageIter)
   {
      const Classifier::Stage & stage = *stageIter;

      for (std::vector<Classifier::SelectionResult>::const_iterator selectionResultIter = stage.stagedClassifier.begin();
            selectionResultIter != stage.stagedClassifier.end();
            ++selectionResultIter)
      {
         const Classifier::SelectionResult & selectionResult = *selectionResultIter;
         xMin = (selectionResult.x < xMin) ? selectionResult.x : xMin;
         yMin = (selectionResult.y < yMin) ? selectionResult.y : yMin;

         const FeatureType & featureType = featureTypes[selectionResult.classifierTypeIdx];
         const uint32_t xRight  = selectionResult.x + (featureType.mFeatureWidth  * featureType.mRect.width);
         const uint32_t yBottom = selectionResult.y + (featureType.mFeatureHeight * featureType.mRect.height);

         xMax = (xRight  > xMax) ? xRight  : xMax;
         yMax = (yBottom > yMax) ? yBottom : yMax;
      }
   }

   printf("sizeStrongClassifier: xMin:%d, yMin:%d, xMax:%d, yMax:%d\n", xMin, yMin, xMax, yMax);
}

void Classifier::scaleStrongClassifier(
         const double scale,
         const std::vector<Classifier::Stage> & strongClassifier,
         const FeatureTypes & featureTypes,
         std::vector<Classifier::Stage> & scaledStrongClassifier,
         FeatureTypes & scaledFeatureTypes)
{
   scaledStrongClassifier = strongClassifier;
   scaledFeatureTypes = featureTypes;
   assert(scaledStrongClassifier.size() == strongClassifier.size());

   for (std::vector<Classifier::Stage>::iterator stageIter = scaledStrongClassifier.begin();
        stageIter != scaledStrongClassifier.end();
        ++stageIter)
   {
      Classifier::Stage & stage = *stageIter;

      for (std::vector<Classifier::SelectionResult>::iterator selectionResultIter = stage.stagedClassifier.begin();
            selectionResultIter != stage.stagedClassifier.end();
            ++selectionResultIter)
      {
         Classifier::SelectionResult & selectionResult = *selectionResultIter;
         selectionResult.x         = static_cast<uint32_t>(static_cast<double>(selectionResult.x) * scale);
         selectionResult.y         = static_cast<uint32_t>(static_cast<double>(selectionResult.y) * scale);
         selectionResult.threshold = static_cast<int32_t>(static_cast<double>(selectionResult.threshold) * pow(scale, 2));
      }
   }

   for (std::vector<FeatureType>::iterator featureTypeIter = scaledFeatureTypes.begin();
        featureTypeIter != scaledFeatureTypes.end();
        ++featureTypeIter)
   {
      FeatureType & featureType = *featureTypeIter;
      featureType.setRect(
            static_cast<uint32_t>(static_cast<double>(featureType.mRect.width) * scale),
            static_cast<uint32_t>(static_cast<double>(featureType.mRect.height) * scale),
            featureType.mRect.type);
   }
}

__device__ __forceinline__ void detectStrongClassifierAtPoint(
      const int32_t * const integralImage,
      const uint32_t imageWidth,
      const uint32_t imageHeight,
      const uint32_t step,
      const uint32_t x,
      const uint32_t y,
      const uint8_t * const allClassifierData,
      const GpuStrongClassifier::Stage * const stages,
      const uint32_t stageCount,
      bool & detected,
      double & hSum
      )
{
   detected = false;

   // for each stage
   for (uint32_t stageIdx = 0; stageIdx < stageCount; ++stageIdx)
   {
      hSum = 0.0;
      const GpuStrongClassifier::Stage & stage = stages[stageIdx];

      // for each classifier in stage
      for (uint32_t classifierIdx = 0;  classifierIdx < stage.mClassifierCount; ++classifierIdx)
      {
         const Classifier::SelectionResult & classifierDescription = stage.mSelectionResults[classifierIdx];
         const double beta = stage.mBetas[classifierIdx];

         double alpha = 40.0;

         if  (beta != 0.0)
             alpha = log(1.0/beta);

         // get all classifier of one type - here we have only one
         uint32_t featureHeight;
         uint32_t featureWidth;
         uint32_t classifierCount;
         const uint8_t * classifiers = NULL;

         Classifier::getClassifier(
               allClassifierData,
               classifierDescription.classifierTypeIdx,
               classifierCount,
               &classifiers,
               featureWidth,
               featureHeight);

         assert(classifiers);

         uint32_t rectWidth;
         uint32_t rectHeight;
         const uint8_t * singleClassifier = NULL;

         Classifier::getClassifierScale(
               classifiers,
               classifierDescription.classifierIdx,
               &singleClassifier,
               rectWidth,
               rectHeight);

         assert(singleClassifier);

         const uint32_t classifierLeftPoint = x + classifierDescription.x;
         const uint32_t classifierUpperPoint = y + classifierDescription.y;
         const uint32_t classifierRightPoint = classifierLeftPoint + rectWidth * featureWidth;
         const uint32_t classifierBottomPoint = classifierUpperPoint + rectHeight * featureHeight;

         const bool outOfRange = ((classifierRightPoint <= imageWidth)
               && (classifierBottomPoint <= imageHeight)) ? false : true;

         int32_t featureValue = INT_MAX;

/*TODO: debug message
         if (threadIdx.x == 0 && blockIdx.x == 0)
         {
            printf("stage:%d featureWidth:%d, featureHeight:%d, rectWidth:%d, rechtHeight:%d typeIdx:%d outOfRange:%d\n",
                  stageIdx, featureWidth, featureHeight, rectWidth, rectHeight, classifierDescription.classifierTypeIdx, outOfRange);
            printf("classifierLeftPoint:%d classifierUpperPoint:%d classifierRightPoint:%d classifierBottomPoint:%d\n",
                  classifierLeftPoint, classifierUpperPoint, classifierRightPoint, classifierBottomPoint);
         }
*/

         if (!outOfRange)
         {
            Classifier::getFeatureValue(
                  integralImage,
                  singleClassifier,
                  // FIXME check this
                  step,
                  classifierLeftPoint,
                  classifierUpperPoint,
                  rectWidth, rectHeight,
                  featureWidth, featureHeight,
                  featureValue);
            const int32_t h = (classifierDescription.polarity * featureValue) < (classifierDescription.polarity * classifierDescription.threshold) ? 1 : 0;
            hSum += static_cast<double>(h) * alpha;
// FIXME remove this
/*TODO: debug message
if (threadIdx.x == 1 && blockIdx.x == 1)
{
  printf("h(%d) = pol(%d) * val(%d) < pol(%d) * threshold(%d)\n", h, classifierDescription.polarity, featureValue, classifierDescription.polarity, classifierDescription.threshold);
  printf("hSum(%f) += h(%d) * alpha(%f)\n\n", hSum, h, alpha);
}
*/
            //alphaSum += alpha;
         }
         else
         {
            hSum = 0.0;
            detected = false;
            return;
         }
      }

      if (hSum < stage.mStageThreshold)
      {
         hSum = 0.0;
         detected = false;
         return;
      }
      else
      {
         detected = true;
      }
   }
}


__global__ void detectStrongClassifierGpu(
      cv::gpu::PtrStepSz<int32_t> integralImage,
      const uint32_t imageWidth,
      const uint32_t imageHeight,
      const uint8_t * const allClassifierData,
      const GpuStrongClassifier::Stage * const stages,
      const uint32_t stageCount,
      double * results)
{
   //const uint8_t * const allClassifierData = &g_FeatureData[0];
   assert(allClassifierData);
   assert(stages);
   assert(results);

   const uint32_t pixelCount = imageWidth * imageHeight;
   const uint32_t pixelIdx =  blockIdx.x * blockDim.x + threadIdx.x;

   if (!(pixelCount > pixelIdx))
   {
      return;
   }

   /*
   const uint32_t y = pixelIdx / integralImage.cols;
   const uint32_t x = pixelIdx - y * integralImage.cols;
   */

   const uint32_t y = pixelIdx / imageWidth;
   const uint32_t x = pixelIdx - y * imageWidth;

// FIXME remove this. Just for debugging
//if (x != 684 || y != 59 )
//   return;

   const int32_t * integralImageData = (int32_t *)(integralImage.data);
   bool detected = false;
   double hSum = 0.0;

   detectStrongClassifierAtPoint(
         integralImageData,
         imageWidth,
         imageHeight,
         integralImage.step / sizeof(uint32_t),
         x, y,
         allClassifierData,
         stages,
         stageCount,
         detected,
         hSum
         );

// FIXME remove this
//printf("Stage Threshold %f\n", stage.mStageThreshold);
//printf("Stage %d done x:%d y:%d\n\n\n",stageIdx,x,y);


   //printf("Match x:%d y:%d\n",x,y);
   results[pixelIdx] = hSum;
}

__global__ void detectStrongClassifierOnImageSetGpu(
      const int32_t * const integralImages,
      const uint32_t startImageIdx,
      const uint32_t imageCount,
      const uint32_t imageWidth,
      const uint32_t imageHeight,
      const uint8_t * const allClassifierData,
      const GpuStrongClassifier::Stage * const stages,
      const uint32_t stageCount,
      bool * allDetected)
{
   assert(integralImages);
   assert(allClassifierData);
   assert(stages);
   assert(allDetected);

   const uint32_t imageIdx =  blockIdx.x * blockDim.x + threadIdx.x;

   if (!(imageIdx < imageCount))
   {
      return;
   }

   const uint32_t pixelCountPerImage = imageHeight * imageWidth;
   const int32_t * const  integralImageData = integralImages + pixelCountPerImage * (imageIdx + startImageIdx);
   bool detected = false;
   double hSum = 0.0;

   detectStrongClassifierAtPoint(
         integralImageData,
         imageWidth,
         imageHeight,
         imageWidth,
         0, 0,
         allClassifierData,
         stages,
         stageCount,
         detected,
         hSum
         );

   allDetected[imageIdx] = detected;
}

bool Classifier::detectStrongClassifier(
      const std::vector<Classifier::Stage> & strongClassifier,
      FeatureTypes & featureTypes,
      const cv::gpu::GpuMat & gpuIntegralImage,
      std::vector<Classifier::ClassificationResult> & results
      )
{
   bool detected = false;
   double * resultsPtr = NULL;
   const uint32_t pixelCount = gpuIntegralImage.cols * gpuIntegralImage.rows;

   uint32_t strongClassifierXmin;
   uint32_t strongClassifierYmin;
   uint32_t strongClassifierXmax;
   uint32_t strongClassifierYmax;
   Classifier::sizeStrongClassifier(strongClassifier, featureTypes, strongClassifierXmin, strongClassifierYmin, strongClassifierXmax, strongClassifierYmax);

   CUDA_CHECK_RETURN(hipMalloc(
         &resultsPtr,
         sizeof(double) * pixelCount)
         );

   const GpuStrongClassifier gpuStrongClassifier(strongClassifier);
   const uint32_t threadCount = 256;
   const uint32_t blockCount = (pixelCount + threadCount - 1) / threadCount;

   uint8_t * gpuFeatureData = FeatureTypes::getConstantFeatureData();

   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipEventRecord(start);

   detectStrongClassifierGpu<<<blockCount, threadCount>>>(
         gpuIntegralImage,
         gpuIntegralImage.cols,
         gpuIntegralImage.rows,
         gpuFeatureData,
   //      featureTypes.getGpuData(),
         gpuStrongClassifier.getGpuStages(),
         gpuStrongClassifier.mStagesCount,
         resultsPtr);

   CUDA_CHECK_RETURN(hipPeekAtLastError());
   CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
   CUDA_CHECK_RETURN(hipGetLastError());

   hipEventRecord(stop);
   hipEventSynchronize(stop);

   dumpElapsedTime("detectStrongClassifier:", start, stop);
   dumpFreeMemory("detectStrongClassifier:");


   // DEBUG
   double * hostResult = new double[pixelCount];
   CUDA_CHECK_RETURN(hipMemcpy(hostResult, resultsPtr, sizeof(double) * pixelCount, hipMemcpyDeviceToHost));

   for (uint32_t i = 0; i < pixelCount; ++i)
   {
      if (hostResult[i] > 0.0)
      {
         uint32_t y = i / gpuIntegralImage.cols;
         uint32_t x = i - gpuIntegralImage.cols * y;
         // std::cout << "detectStrongClassifier: at x:" << x << " y:" << y << std::endl;
         Classifier::ClassificationResult classificationResult;
         classificationResult.x = x; // x + strongClassifierXmin;
         classificationResult.y = y; // y + strongClassifierYmin;
         classificationResult.height = strongClassifierYmax - strongClassifierYmin;
         classificationResult.width = strongClassifierXmax - strongClassifierXmin;
         classificationResult.strength = hostResult[i];
         addUniqueResult(classificationResult, results);
      }
   }
   ////////////////

   CUDA_CHECK_RETURN(hipFree(resultsPtr));
   return detected;
}

void Classifier::detectStrongClassifierOnImageSet(
      const std::vector<Classifier::Stage> & strongClassifier,
      FeatureTypes & featureTypes,
      const int32_t * const gpuIntegralImages,
      const uint32_t startImageIdx,
      const uint32_t imageCount,
      const uint32_t imageWidth,
      const uint32_t imageHeight,
      bool * results
      )
{
#ifdef DEBUG
         std::cout << "Debug: detectStrongClassifierOnImageSet startImageIdx:" << startImageIdx
               << ", imageCount:" << imageCount
               << ", imageWidth:" << imageWidth
               << ", imageHeight:" << imageHeight
               << ", stageCount:" << strongClassifier.size()
               << std::endl;
#endif

   bool * resultsGpu = NULL;

   CUDA_CHECK_RETURN(hipMalloc(
         &resultsGpu,
         sizeof(bool) * imageCount)
         );

   const GpuStrongClassifier gpuStrongClassifier(strongClassifier);
   const uint32_t threadCount = 256;
   const uint32_t blockCount = (imageCount + threadCount - 1) / threadCount;
   uint8_t * gpuFeatureData = FeatureTypes::getConstantFeatureData();

   hipEvent_t start;
   hipEvent_t stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);

   detectStrongClassifierOnImageSetGpu<<<blockCount, threadCount>>>(
         gpuIntegralImages,
         startImageIdx,
         imageCount,
         imageWidth,
         imageHeight,
         gpuFeatureData,
         gpuStrongClassifier.getGpuStages(),
         gpuStrongClassifier.mStagesCount,
         resultsGpu);

   CUDA_CHECK_RETURN(hipPeekAtLastError());
   CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
   CUDA_CHECK_RETURN(hipGetLastError());

   hipEventRecord(stop);
   hipEventSynchronize(stop);

   dumpElapsedTime("detectStrongClassifierOnImageSet:", start, stop);
   dumpFreeMemory("detectStrongClassifierOnImageSet:");

   CUDA_CHECK_RETURN(
         hipMemcpy(
               results,
               resultsGpu,
               sizeof(bool) * imageCount,
               hipMemcpyDeviceToHost));

   CUDA_CHECK_RETURN(hipFree(resultsGpu));
}


void Classifier::addUniqueResult(const Classifier::ClassificationResult & newResult, std::vector<Classifier::ClassificationResult> & results)
{
   bool isUnique = true;

   // https://stackoverflow.com/questions/9324339/how-much-do-two-rectangles-overlap
   // SI = Max(0, Max(XA2, XB2) - Min(XA1, XB1)) * Max(0, Max(YA2, YB2) - Min(YA1, YB1))
   // SU = SA + SB - SI
   // ratio = SI / SU
   const int32_t xa1 = newResult.x;
   const int32_t xa2 = xa1 + newResult.width;

   const int32_t ya1 = newResult.y;
   const int32_t ya2 = ya1 + newResult.height;

   const int32_t areaA = newResult.width * newResult.height;

   for (std::vector<Classifier::ClassificationResult>::const_iterator resultIter = results.begin();
        resultIter != results.end();
        ++resultIter)
   {
      const int32_t xb1 = (*resultIter).x;
      const int32_t xb2 = xb1 + (*resultIter).width;

      const int32_t yb1 = (*resultIter).y;
      const int32_t yb2 = yb1 + (*resultIter).height;

      const int32_t areaB = (*resultIter).width * (*resultIter).height;

      const int32_t areaIntersect = max(0, max(xa2, xb2) - min(xa1, xb1)) * max(0, max(ya2, yb2) - min(ya1, yb1));
      const int32_t areaUnion = areaA + areaB - areaIntersect;

      double ratio = 1.0;

      if (areaUnion != 0.0)
      {
         ratio = areaIntersect / areaUnion;
      }

      if (ratio > 0.5)
      {
         isUnique = false;
         break;
      }
   }

   if (isUnique)
   {
      results.push_back(newResult);
   }
}
/*
texture<int32_t, 2> & Classifier::getTexIntegralImage()
{
   return texIntegralImage;
}
*/
