/*
 *
 *  Created on: May 17, 2017
 *      Author: Mario Lüder
 *
 */


#include "Classifier.cuh"
#include "GpuStrongClassifier.cuh"
#include "defines.cuh"

GpuStrongClassifier::GpuStrongClassifier(const std::vector<Classifier::Stage> & stagedClassifer)
: mStagesCount(stagedClassifer.size())
, mStages(NULL)
{
   CUDA_CHECK_RETURN(
         hipMalloc(&mStages, sizeof(Stage) * mStagesCount));

   for (uint32_t i = 0; i < mStagesCount; ++i)
   {
      const Classifier::Stage & stage = stagedClassifer[i];
      Stage gpuStage;

      // create space for betas
      double * gpuBetaPtr;

      CUDA_CHECK_RETURN(
            hipMalloc(&gpuBetaPtr, sizeof(double) * stage.betas.size()));
      CUDA_CHECK_RETURN(
            hipMemcpy(
                  gpuBetaPtr,
                  &stage.betas[0],
                  sizeof(double) * stage.betas.size(),
                  hipMemcpyHostToDevice));

      Classifier::SelectionResult * gpuSelectionResultPtr = NULL;

      CUDA_CHECK_RETURN(
            hipMalloc(
                  &gpuSelectionResultPtr,
                  sizeof(Classifier::SelectionResult) * stage.stagedClassifier.size()));

      CUDA_CHECK_RETURN(
            hipMemcpy(
                  gpuSelectionResultPtr,
                  &stage.stagedClassifier[0],
                  sizeof(Classifier::SelectionResult) * stage.stagedClassifier.size(),
                  hipMemcpyHostToDevice));

      gpuStage.mBetas = gpuBetaPtr;
      gpuStage.mSelectionResults = gpuSelectionResultPtr;
      gpuStage.mClassifierCount = stage.stagedClassifier.size();
      gpuStage.mStageThreshold = stage.stageThreshold;

      CUDA_CHECK_RETURN(
            hipMemcpy(
                  &mStages[i],
                  &gpuStage,
                  sizeof(Stage),
                  hipMemcpyHostToDevice));

   }
}

GpuStrongClassifier::~GpuStrongClassifier()
{
   for (uint32_t i = 0; i < mStagesCount; ++i)
   {
      Stage cpuStage;

      CUDA_CHECK_RETURN(
            hipMemcpy(
                  &cpuStage,
                  &mStages[i],
                  sizeof(Stage),
                  hipMemcpyDeviceToHost));

      CUDA_CHECK_RETURN(hipFree(cpuStage.mBetas));
      CUDA_CHECK_RETURN(hipFree(cpuStage.mSelectionResults));
   }

   hipFree(mStages);
}
